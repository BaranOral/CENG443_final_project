#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 100

__global__ void bubbleSort(float *d_arr) {
  // Compare all pairs of elements
  for (int i = 0; i < N - 1; i++) {
    // Get the index of the current element to be sorted
    int j = threadIdx.x + i;

    // Compare the current element with the next element
    if (d_arr[j] > d_arr[j + 1]) {
      // Swap the elements if the current element is greater than the next
      float temp = d_arr[j];
      d_arr[j] = d_arr[j + 1];
      d_arr[j + 1] = temp;
    }
  }
}

__global__ void selectionSort(float *d_arr) {
  // Get the index of the current element to be sorted
  int i = threadIdx.x;

  // Find the minimum element in the array
  int minIndex = i;
  for (int j = i + 1; j < N; j++) {
    if (d_arr[j] < d_arr[minIndex]) {
      minIndex = j;
    }
  }

  // Swap the current element with the minimum element
  float temp = d_arr[i];
  d_arr[i] = d_arr[minIndex];
  d_arr[minIndex] = temp;
}

__global__ void insertionSort(float *d_arr) {
  int j;
  float temp;

  // Get the index of the current thread
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= N) {
    return;  // Out of bounds, return early
  }

  temp = d_arr[idx];
  j = idx - 1;
  while (j >= 0 && d_arr[j] > temp) {
    d_arr[j + 1] = d_arr[j];
    j--;
  }
  d_arr[j + 1] = temp;
}

__host__ float* generateRandomElements(){
    float constant = 1.0f;
    float *arr;
    arr = (float*)malloc((N + 1) * sizeof(float));  
    if (arr == NULL) {
      printf("Run out of memmory!\n");
      exit(1);
    }

    for (int i = 0; i < N; i++) {
      arr[i + 1] = ((float)rand() / RAND_MAX) * constant; //generate random float element for array
    }
    return &arr[1];
}

void printArray(float *array, int size){
    // Iterate over the elements of the array
    for (int i = 0; i < size; i++)
    {
        // Print the element
        printf("%f ", array[i]);
}
     printf("\n");

}

__global__ void bubbleSortWithSharedMemory(float *d_arr) {
  // Declare shared memory array
  __shared__ float s_arr[N];

  // Load a portion of the data from global memory into shared memory
  int i = threadIdx.x;
  s_arr[i] = d_arr[i];
  __syncthreads();

  // Sort the data in shared memory
  
    for (int i = 0; i < N - 1; i++) {
    // Get the index of the current element to be sorted
    int j = threadIdx.x + i;

    // Compare the current element with the next element
    if (s_arr[j] > s_arr[j + 1]) {
      // Swap the elements if the current element is greater than the next
      float temp = s_arr[j];
      s_arr[j] = s_arr[j+1];
      s_arr[j+1] = temp;
      
    }
    // each thread writes one element back to global memory
    
  }
    __syncthreads(); // synchronize threads again before writing back to global memory
    d_arr[i] = s_arr[i];
}
int main(void) {
  // Create host array
  
  float *h_arr, *d_arr;

  h_arr = generateRandomElements();

  // Allocate memory on the device
  hipMalloc(&d_arr, N * sizeof(float));

  // Copy host array to device
  hipMemcpy(d_arr, h_arr, N * sizeof(float), hipMemcpyHostToDevice);

  // Launch bubbleSort kernel
  bubbleSortWithSharedMemory<<<1, N>>>(d_arr);

  // Copy the sorted array back to the host
  hipMemcpy(h_arr, d_arr, N * sizeof(float), hipMemcpyDeviceToHost);

  printArray(h_arr, N);
 
  // Free memory on the device
  hipFree(d_arr);

}
